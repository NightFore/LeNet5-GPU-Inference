// Part1-MatrixOperations.cu

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>


// Kernel prototypes
__global__ void cudaMatrixAdd(float* M1, float* M2, float* Mout, int n, int p);
__global__ void cudaMatrixMult(float* M1, float* M2, float* Mout, int n);

// Function prototypes
void MatrixInit(float *M, int n, int p);
void MatrixPrint(float *M, int n, int p);
void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p);
void MatrixMult(float *M1, float *M2, float *Mout, int n);


// CUDA kernel to add two matrices
__global__ void cudaMatrixAdd(float* M1, float* M2, float* Mout, int n, int p) {
    // Calculate the row and column indices
    int row = blockIdx.x;
    int col = threadIdx.x;

    // Ensure we don't go out of bounds
    if (row < n && col < p) {
        // Perform the matrix addition
        Mout[row * p + col] = M1[row * p + col] + M2[row * p + col];
    }
}

// CUDA kernel to multiply two NxN matrices
__global__ void cudaMatrixMult(float* M1, float* M2, float* Mout, int n) {
    // Calculate the row and column indices
    int row = blockIdx.x;
    int col = threadIdx.x;

    // Ensure we don't go out of bounds
    if (row < n && col < n) {
        // Initialize the output element to 0
        Mout[row * n + col] = 0.0f;

        // Perform the matrix multiplication
        for (int k = 0; k < n; k++) {
            Mout[row * n + col] += M1[row * n + k] * M2[k * n + col];
        }
    }
}

// Function to initialize the matrix
void MatrixInit(float* M, int n, int p) {
    // Iterate through each element of the matrices
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            // Assign a random value between -1 and 1 to each element
            M[i * p + j] = ((float)rand() / RAND_MAX) * 2.0f - 1.0f;
        }
    }
}

// Function to print the matrix
void MatrixPrint(float* M, int n, int p) {
    // Iterate through each element of the matrices
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            // Print each element of the matrix with 2 decimal places
            printf("%6.2f ", M[i * p + j]);
        }
        printf("\n");
    }
}

// Function to add two matrices (CPU version)
void MatrixAdd(float* M1, float* M2, float* Mout, int n, int p) {
    // Iterate through each element of the matrices
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            // Perform the matrix addition
            Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
        }
    }
}

// Function to multiply two NxN matrices (CPU version)
void MatrixMult(float* M1, float* M2, float* Mout, int n) {
    // Iterate through each element of the matrices
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            // Initialize the output element to 0
            Mout[i * n + j] = 0;

            // Perform the matrix multiplication
            for (int k = 0; k < n; k++) {
                Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
            }
        }
    }
}

int main(int argc, char *argv[]) {
    // Set default matrix dimensions and print option
    int n = 1000, p = 1000;
    bool printMatrices = false;

    // Override default values based on command line arguments (if provided)
    if (argc > 1) n = atoi(argv[1]);
    if (argc > 2) p = atoi(argv[2]);
    if (argc > 3 && argv[3][0] == '1') printMatrices = true;

    // Allocate memory for matrices
    float *matrix1 = (float*) malloc(n * p * sizeof(float));
    float *matrix2 = (float*) malloc(n * p * sizeof(float));
    float *resultMatrix = (float*) malloc(n * p * sizeof(float));

    // Declare GPU memory pointers
    float *d_M1, *d_M2, *d_Mout;

    // Timing variables
    clock_t cpuStart, cpuEnd;
    hipEvent_t gpuStart, gpuStop;
    float cpuTime, gpuTime;
    hipEventCreate(&gpuStart);
    hipEventCreate(&gpuStop);

    printf("---------- Matrix Initialization ----------\n");
    // Initialize matrices with random values
    MatrixInit(matrix1, n, p);
    printf("Matrix 1 initialized\n");

    MatrixInit(matrix2, n, p);
    printf("Matrix 2 initialized\n");

    // Print the result
    if (printMatrices) {
        MatrixPrint(matrix1, n, p);
        MatrixPrint(matrix2, n, p);
    }

    printf("\n---------- Matrix Addition (CPU) ----------\n");
    cpuStart = clock();
    MatrixAdd(matrix1, matrix2, resultMatrix, n, p);
    cpuEnd = clock();
    cpuTime = ((double)(cpuEnd - cpuStart)) / CLOCKS_PER_SEC;
    printf("Time taken for CPU Addition: %f seconds\n", cpuTime);

    printf("\n---------- Matrix Addition (GPU) ----------\n");
    // Allocate memory on the GPU
    hipMalloc((void**)&d_M1, n * p * sizeof(float));
    hipMalloc((void**)&d_M2, n * p * sizeof(float));
    hipMalloc((void**)&d_Mout, n * p * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_M1, matrix1, n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M2, matrix2, n * p * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel and measure the time
    hipEventRecord(gpuStart);
    cudaMatrixAdd<<<n, p>>>(d_M1, d_M2, d_Mout, n, p);
    hipDeviceSynchronize();
    hipEventRecord(gpuStop);
    hipEventSynchronize(gpuStop);
    hipEventElapsedTime(&gpuTime, gpuStart, gpuStop);

    // Copy the result from device to host
    hipMemcpy(resultMatrix, d_Mout, n * p * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_M1);
    hipFree(d_M2);
    hipFree(d_Mout);

    // Print the result
    printf("Time taken for GPU Addition: %f seconds\n", gpuTime / 1000);
    if (printMatrices) {
        printf("\nResult of Matrix Addition:\n");
    	MatrixPrint(resultMatrix, n, p);
    }

    printf("\n---------- Matrix Multiplication (CPU) ----------\n");
    cpuStart = clock();
    MatrixMult(matrix1, matrix2, resultMatrix, n);
    cpuEnd = clock();
    cpuTime = ((double)(cpuEnd - cpuStart)) / CLOCKS_PER_SEC;
    printf("Time taken for CPU Multiplication: %f seconds\n", cpuTime);

    printf("\n---------- Multiplication Part (GPU) ----------\n");
    // Allocate memory on the GPU
    hipMalloc((void**)&d_M1, n * p * sizeof(float));
    hipMalloc((void**)&d_M2, n * p * sizeof(float));
    hipMalloc((void**)&d_Mout, n * p * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_M1, matrix1, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M2, matrix2, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel and measure the time
    hipEventRecord(gpuStart);
    cudaMatrixMult<<<n, n>>>(d_M1, d_M2, d_Mout, n);
    hipDeviceSynchronize();
    hipEventRecord(gpuStop);
    hipEventSynchronize(gpuStop);
    hipEventElapsedTime(&gpuTime, gpuStart, gpuStop);

    // Copy the result from device to host
    hipMemcpy(resultMatrix, d_Mout, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_M1);
    hipFree(d_M2);
    hipFree(d_Mout);

    // Print the result
    printf("Time taken for GPU Multiplication: %f seconds\n", gpuTime / 1000);
    if (printMatrices) {
        printf("\nResult of Matrix Multiplication:\n");
    	MatrixPrint(resultMatrix, n, n);
    }

    // Free the allocated memory
    free(matrix1);
    free(matrix2);
    free(resultMatrix);

    return 0;
}
